#include "cuda_gemm.cuh"
#include <hipsparse.h>

namespace cuda {
    void gemm_4_AX(const CSRGraph_t &A_csr, // raw_graph : (v_num * v_num)
                const f32*      X,       // X : (v_num * dim)
                f32*            Y,       // Y : (v_num * dim)
                const uint      dim,
                const uint      v_num) 
    {
        // 1. 获取矩阵维度信息
        const int m = v_num;
        const int n = dim;
        const int k = v_num;
        const int nnz = A_csr.data.size();

        // 2. 在设备(GPU)上分配内存
        int* d_A_row_pointers;
        int* d_A_indices;
        f32* d_A_data;
        f32* d_X;
        f32* d_Y;

        hipMalloc(&d_A_row_pointers, A_csr.index_pointers.size() * sizeof(int));
        hipMalloc(&d_A_indices, nnz * sizeof(int));
        hipMalloc(&d_A_data, nnz * sizeof(f32));
        hipMalloc(&d_X, (size_t)k * n * sizeof(f32));
        hipMalloc(&d_Y, (size_t)m * n * sizeof(f32));

        // 3. 将数据从主机(CPU)拷贝到设备(GPU)
        hipMemcpy(d_A_row_pointers, A_csr.index_pointers.data(), A_csr.index_pointers.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_A_indices, A_csr.indices.data(), nnz * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_A_data, A_csr.data.data(), nnz * sizeof(f32), hipMemcpyHostToDevice);
        hipMemcpy(d_X, X, (size_t)k * n * sizeof(f32), hipMemcpyHostToDevice);
        
        // 4. cuSPARSE 计算
        hipsparseHandle_t handle;
        hipsparseCreate(&handle);

        // 创建矩阵描述符
        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;

        // 描述稀疏矩阵 A (m x k)
        hipsparseCreateCsr(&matA, m, k, nnz,
                        d_A_row_pointers, d_A_indices, d_A_data,
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
        
        // 描述稠密矩阵 B (X) (k x n)，行主序
        hipsparseCreateDnMat(&matB, k, n, n, d_X, HIP_R_32F, HIPSPARSE_ORDER_ROW);
        
        // 描述稠密矩阵 C (Y) (m x n)，行主序
        hipsparseCreateDnMat(&matC, m, n, n, d_Y, HIP_R_32F, HIPSPARSE_ORDER_ROW);

        // 执行 SpMM (Y = 1.0 * A * X + 0.0 * Y)
        const f32 alpha = 1.0f;
        const f32 beta  = 0.0f;
        void* d_buffer = nullptr;
        size_t buffer_size = 0;

        hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, &buffer_size);
        hipMalloc(&d_buffer, buffer_size);

        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha, matA, matB, &beta, matC, HIP_R_32F,
                    HIPSPARSE_SPMM_ALG_DEFAULT, d_buffer);

        // 5. 将结果从设备拷贝回主机
        hipMemcpy(Y, d_Y, (size_t)m * n * sizeof(f32), hipMemcpyDeviceToHost);
        
        // 6. 释放所有资源
        hipFree(d_buffer);
        hipFree(d_A_row_pointers);
        hipFree(d_A_indices);
        hipFree(d_A_data);
        hipFree(d_X);
        hipFree(d_Y);
        
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);
        hipsparseDestroy(handle);
    }
} // namespace cuda
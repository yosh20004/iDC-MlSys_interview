#include "hip/hip_runtime.h"
#include "prepare.h"
#include "kernel.cuh"
#include <hip/hip_runtime.h>
#include <chrono>

typedef std::chrono::time_point<std::chrono::steady_clock> TimePoint;

int main() {
    const auto raw_graph = make_raw_graph(v_num); // A:(v_num, v_num)
    const cpu::CSRGraph_t csrA = RawGraph2CSR(raw_graph, v_num);

    const auto X = alloc<f32, true>(v_num * dim); // X:(v_num, dim)
    const uint nnz = csrA.data.size();

    f32* d_X = nullptr;
    f32* d_Y = nullptr;
    int* d_col_indices = nullptr;
    int* d_row_indices = nullptr;
    f32* d_data = nullptr;

    hipMalloc((void**)&d_X, v_num * dim * sizeof(f32));
    hipMalloc((void**)&d_col_indices, csrA.col_indices.size() * sizeof(int));
    hipMalloc((void**)&d_row_indices, csrA.row_indices.size() * sizeof(int));
    hipMalloc((void**)&d_data, csrA.data.size() * sizeof(f32));
    hipMalloc((void**)&d_Y, v_num * dim * sizeof(f32));

    hipMemcpy(d_X, X, v_num * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, csrA.col_indices.data(), nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_indices, csrA.row_indices.data(), nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, csrA.data.data(), nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 

    cuda::CSRGraph_t d_csrA = {d_col_indices, 
                               d_row_indices,
                               d_data};

    const dim3 BlockSize = 512;
    const dim3 gridSize = dim3{ nnz,
                                1,
                                (dim + BlockSize.x - 1) / BlockSize.x};
    
    hipDeviceSynchronize();
    TimePoint t1 = std::chrono::steady_clock::now();
    for (int i = 0; i < TIMES; ++i) {
        hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
        cuda::gemm_4_AX_v2<<<gridSize, BlockSize>>>(d_csrA, 
                                                    d_X,
                                                    d_Y,
                                                    v_num,
                                                    dim           );

        hipDeviceSynchronize();
    }
    TimePoint t2 = std::chrono::steady_clock::now();

    hipError_t err_msg = hipGetLastError();
    if (err_msg != hipSuccess) {
        printf("err : %s", hipGetErrorString(err_msg));
    }

    f32* h_Y;
    f32* correct_Y;
    
    h_Y = alloc<f32>(v_num * dim);
    correct_Y = alloc<f32>(v_num * dim);
    cpu::gemm_4_AX(csrA, X, correct_Y, dim, v_num);
    
    double err = 0;
    hipMemcpy(h_Y, d_Y, v_num * dim * sizeof(f32), hipMemcpyDeviceToHost);
    for (uint i = 0; i < v_num * dim; ++i) {
        err = err > std::abs(h_Y[i] - correct_Y[i]) ? 
              err : std::abs(h_Y[i] - correct_Y[i]);
    }
    std::printf("max diff = %.3e\n", err);
    std::printf("avg_time : %.3f ms\n",
                std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() / (double)TIMES);
}
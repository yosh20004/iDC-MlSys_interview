#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "prepare.h"


namespace cuda {
    __global__ void gemm_4_AX_v1(const CSRGraph_t d_csrA, // v_num * v_num 
                                 const f32* X,            // v_num * dim
                                 f32* Y,                  // v_num * dim
                                 const uint v_num,
                                 const uint dim) 
    {   
        const uint global_id = blockIdx.x;
        const uint A_col_index = d_csrA.col_indices[global_id];
        const uint A_row_index = d_csrA.row_indices[global_id];
        const uint X_row_index = A_col_index;
        const f32 A_ele = d_csrA.data[global_id];

        const uint tid = threadIdx.x + blockDim.x * blockIdx.z;
        const uint X_col_index = tid;
        const uint Y_row_index = A_row_index;
        const uint Y_col_index = X_col_index;

        if (tid < dim) {
            const f32 X_ele = X[X_row_index * dim + X_col_index]; 
            const f32 tmp = A_ele * X_ele;
            atomicAdd(&Y[Y_row_index * dim + Y_col_index], tmp);
        } 
    }
}


namespace cuda {
    // this kernel requires BlockSize == 512
    __global__ void gemm_4_AX_v2(const CSRGraph_t d_csrA, // v_num * v_num 
                                 const f32* X,            // v_num * dim
                                 f32* Y,                  // v_num * dim
                                 const uint v_num,
                                 const uint dim)    
    {   
        const uint global_id = blockIdx.x;
        const uint A_col_index = d_csrA.col_indices[global_id];
        const uint A_row_index = d_csrA.row_indices[global_id];
        const uint X_row_index = A_col_index;
        const f32 A_ele = d_csrA.data[global_id];

        const uint tid = threadIdx.x + blockDim.x * blockIdx.z;
        const uint X_col_index = tid;
        const uint Y_row_index = A_row_index;

        __shared__ f32 Y_local[512];
        if (tid >= dim) 
            return;

        const f32 X_ele = X[X_row_index * dim + X_col_index]; 
        const f32 tmp = A_ele * X_ele;
        Y_local[threadIdx.x] = tmp;

        // 只有一个线程负责访存，速度太慢了
        if (threadIdx.x == 0) {
            for (uint i = 0; i < blockDim.x; ++i) {
                atomicAdd(&Y[Y_row_index * dim + i + blockDim.x * blockIdx.z], Y_local[i]);
            }
        }
    }
}


namespace cuda {
    template<typename T, typename Op>
    __device__ T block_reduce(T* smem, Op op)
    {
        int tid = threadIdx.x;
        T val   = smem[tid];
        
        for (int s = blockDim.x / 2; s > 32; s >>= 1) {
            if (tid < s) smem[tid] = op(smem[tid], smem[tid + s]);
            __syncthreads();
        }

        if (tid < 32) val = (blockDim.x > 32) ? smem[tid] : val;
        for (int offset = 16; offset > 0; offset >>= 1) {
            T other = __shfl_down_sync(0xffffffff, val, offset);
            val = op(val, other);
        }

        return val;
    }


    // this kernel requires BlockSize == 16
    __global__ void gemm_4_AX_v3(const CSRGraph_t d_csrA, // v_num * v_num 
                                 const f32* X,            // v_num * dim
                                 f32* Y,                  // v_num * dim
                                 const uint v_num,
                                 const uint dim)
    {
        const uint Y_row_index = blockIdx.x;
        const uint Y_col_index = blockIdx.y;
        const uint stride = blockDim.x;

        const uint start_index = d_csrA.index_pointers[Y_row_index];
        const uint end_index = d_csrA.index_pointers[Y_row_index + 1];

        __shared__ f32 buffer[16];
        buffer[threadIdx.x] = 0.0f;

        for (uint i = start_index; 
             i < end_index; 
             i += stride) {
            
            const uint local_index = i + threadIdx.x;
            if (local_index >= end_index) 
                continue;
            
            const uint A_col_index = d_csrA.col_indices[local_index];
            const uint X_row_index = A_col_index;
            const uint X_col_index = Y_col_index;

            const f32 A_ele = d_csrA.data[local_index];
            const f32 X_ele = X[X_row_index * dim + X_col_index];

            buffer[threadIdx.x] += A_ele * X_ele;
        }

        float sum = block_reduce(buffer, 
                      [] __device__ (f32 a, f32 b) { return a + b; });

        if (threadIdx.x == 0)
            Y[Y_row_index * dim + Y_col_index] += sum;
    }        
}


namespace cuda {
    __global__ void gemm_4_AX_v4(const CSRGraph_t d_csrA, // v_num * v_num 
                                 const f32* X,            // v_num * dim
                                 f32* Y,                  // v_num * dim
                                 const uint v_num,
                                 const uint dim)
    {
        const uint Y_row_index = blockIdx.x;
        const uint Y_col_index = blockIdx.y * 16 + threadIdx.y;
        const uint stride = blockDim.x;
        const uint Y_line_size = blockDim.y;
        if (Y_col_index >= dim) 
            return;

        const uint start_index = d_csrA.index_pointers[Y_row_index];
        const uint end_index = d_csrA.index_pointers[Y_row_index + 1];
        __shared__ f32 buffer[16][16];
        buffer[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();

        for (uint i = start_index; 
             i < end_index; 
             i += stride) {
            
            const uint local_index = i + threadIdx.x;
            if (local_index >= end_index) 
                continue;
            
            const uint A_col_index = d_csrA.col_indices[local_index];
            const uint X_row_index = A_col_index;
            const uint X_col_index = Y_col_index;

            const f32 A_ele = d_csrA.data[local_index];
            const f32 X_ele = X[X_row_index * dim + X_col_index];

            buffer[threadIdx.y][threadIdx.x] += A_ele * X_ele;
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            f32 sum = buffer[threadIdx.y][0] + 
                      buffer[threadIdx.y][1] + 
                      buffer[threadIdx.y][2] + 
                      buffer[threadIdx.y][3] +
                      buffer[threadIdx.y][4] +
                      buffer[threadIdx.y][5] +
                      buffer[threadIdx.y][6] +
                      buffer[threadIdx.y][7] +
                      buffer[threadIdx.y][8] +
                      buffer[threadIdx.y][9] +
                      buffer[threadIdx.y][10] +
                      buffer[threadIdx.y][11] +
                      buffer[threadIdx.y][12] +
                      buffer[threadIdx.y][13] +
                      buffer[threadIdx.y][14] +
                      buffer[threadIdx.y][15];

            if (Y_col_index < dim) {
                Y[Y_row_index * dim + Y_col_index] = sum;
            }
        }
    }
}



namespace cuda {
    template<>
    void launch_kernel<version::v1>(CSRGraph_t d_csrA,
                                   const uint nnz, 
                                   const f32* d_X,
                                   f32* d_Y,
                                   const uint v_num,
                                   const uint dim)
    {
        const dim3 BlockSize = 512;
        const dim3 gridSize = dim3{ nnz,
                                   1,
                                   (dim + BlockSize.x - 1) / BlockSize.x};
        hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
        for (int i = 0; i < TIMES; ++i) {
            hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
            cuda::gemm_4_AX_v1<<<gridSize, BlockSize>>>(d_csrA, 
                                                       d_X,
                                                       d_Y,
                                                       v_num,
                                                       dim);
            hipDeviceSynchronize();
        }
    }
}



namespace cuda {
    template<>
    void launch_kernel<version::v2>(CSRGraph_t d_csrA,
                                   const uint nnz, 
                                   const f32* d_X,
                                   f32* d_Y,
                                   const uint v_num,
                                   const uint dim)
    {
        const dim3 BlockSize = 512;
        const dim3 gridSize = dim3{ nnz,
                                   1,
                                   (dim + BlockSize.x - 1) / BlockSize.x};
        hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
        for (int i = 0; i < TIMES; ++i) {
            hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
            cuda::gemm_4_AX_v2<<<gridSize, BlockSize>>>(d_csrA, 
                                                       d_X,
                                                       d_Y,
                                                       v_num,
                                                       dim);
            hipDeviceSynchronize();
        }
    }
}



namespace cuda {
    template<>
    void launch_kernel<version::v3>(CSRGraph_t d_csrA,
                                   const uint nnz, 
                                   const f32* d_X,
                                   f32* d_Y,
                                   const uint v_num,
                                   const uint dim)
    {
        const dim3 BlockSize = 16;
        const dim3 gridSize = dim3{v_num,
                                   dim, 
                                   1};
        hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
        for (int i = 0; i < 10; ++i) {
            hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
            cuda::gemm_4_AX_v3<<<gridSize, BlockSize>>>(d_csrA, 
                                                       d_X,
                                                       d_Y,
                                                       v_num,
                                                       dim);
            hipDeviceSynchronize();
        }
    }
}



namespace cuda {
    template<>
    void launch_kernel<version::v4>(CSRGraph_t d_csrA,
                                   const uint nnz, 
                                   const f32* d_X,
                                   f32* d_Y,
                                   const uint v_num,
                                   const uint dim)
    {
        const dim3 BlockSize = dim3{16, 16, 1};
        const dim3 gridSize = dim3{v_num,
                                   (dim + 16 - 1) / 16,
                                   1};
        hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
        for (int i = 0; i < TIMES; ++i) {
            hipMemset(d_Y, 0, v_num * dim * sizeof(float)); 
            cuda::gemm_4_AX_v4<<<gridSize, BlockSize>>>(d_csrA, 
                                                       d_X,
                                                       d_Y,
                                                       v_num,
                                                       dim);
            hipDeviceSynchronize();
        }
    }
}